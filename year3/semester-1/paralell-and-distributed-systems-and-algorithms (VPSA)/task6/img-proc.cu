#include "hip/hip_runtime.h"
//
//  Created by Patricio Bulic, Davor Sluga, UL FRI on 6/6/2022.
//  Copyright © 2022 Patricio Bulic, Davor Sluga UL FRI. All rights reserved.
//

#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 4
#define BLOCK_SIZE 16
//#define CHUNK_SIZE (BLOCK_SIZE + 2)

//***************************************************
// Image sharpening using a 3x3 kernel; 
// Source: https://setosa.io/ev/image-kernels/
//
//      |  0  -1   0 |
// K =  | -1   5  -1 |
//      |  0  -1   0 |
//
//***************************************************

__device__ inline unsigned char getIntensity(const unsigned char *image, int row, int col,
                                             int channel, int height, int width, int cpp)
{
    if (col < 0 || col >= width)
        return 0;
    if (row < 0 || row >= height)
        return 0;
    return image[(row * width + col) * cpp + channel];
}


// CUDA kernel for image sharpening. Each thread computes one output pixel
__global__ void sharpen(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height, const int cpp)
{
    // Get pixel
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        for (int c = 0; c < cpp; c++)
        {
            unsigned char px01 = getIntensity(imageIn, y - 1, x, c, height, width, cpp);
            unsigned char px10 = getIntensity(imageIn, y, x - 1, c, height, width, cpp);
            unsigned char px11 = getIntensity(imageIn, y, x, c, height, width, cpp);
            unsigned char px12 = getIntensity(imageIn, y, x + 1, c, height, width, cpp);
            unsigned char px21 = getIntensity(imageIn, y + 1, x, c, height, width, cpp);

            short pxOut = (5 * px11 - px01 - px10 - px12 - px21);
            pxOut = MIN(pxOut, 255);
            pxOut = MAX(pxOut, 0);
            imageOut[(y * width + x) * cpp + c] = (unsigned char)pxOut;
        }
    }
}


/**
 * Gets intensity for the serial algorithm.
 */
unsigned char getIntensitySingle(const unsigned char *image, int row, int col,
                                             int channel, int height, int width, int cpp)
{
    if (col < 0 || col >= width)
        return 0;
    if (row < 0 || row >= height)
        return 0;
    return image[(row * width + col) * cpp + channel];
}

/**
 * Serially executes the sharpening algorithm.
 */
void sharpenSingle(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height, const int cpp) {
    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) {
            for (int c = 0; c < cpp; c++)
            {
                unsigned char px01 = getIntensitySingle(imageIn, y - 1, x, c, height, width, cpp);
                unsigned char px10 = getIntensitySingle(imageIn, y, x - 1, c, height, width, cpp);
                unsigned char px11 = getIntensitySingle(imageIn, y, x, c, height, width, cpp);
                unsigned char px12 = getIntensitySingle(imageIn, y, x + 1, c, height, width, cpp);
                unsigned char px21 = getIntensitySingle(imageIn, y + 1, x, c, height, width, cpp);
                short pxOut = (5 * px11 - px01 - px10 - px12 - px21);
                pxOut = MIN(pxOut, 255);
                pxOut = MAX(pxOut, 0);
                imageOut[(y * width + x) * cpp + c] = (unsigned char)pxOut;
            }
        }
    }
}

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("USAGE: prog input_image output_image\n");
        exit(EXIT_FAILURE);
    }
    
    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);
    cpp = COLOR_CHANNELS;

    if (h_imageIn == NULL)
    {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(ceil(width / blockSize.x), ceil(height / blockSize.y));

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;



    /*
        Serial algorithm.
    */

    struct timespec timeStart, timeEnd;
    clock_t start_time = clock();
    clock_gettime(CLOCK_REALTIME, &timeStart);

    unsigned char *d_imageInSingle = (unsigned char *)malloc(datasize);
    memcpy(d_imageInSingle, h_imageIn, datasize);
    unsigned char *d_imageOutSingle = (unsigned char *)malloc(datasize);

    sharpenSingle(d_imageInSingle, d_imageOutSingle, width, height, cpp);

    clock_gettime(CLOCK_REALTIME, &timeEnd);
    clock_t end_time = clock();
    double time_taken = ((double)(end_time-start_time))/CLOCKS_PER_SEC; // calculate the elapsed time
    
    double elapsed_time = (timeEnd.tv_sec - timeStart.tv_sec) + (timeEnd.tv_nsec - timeStart.tv_nsec) / 1e9;    // in seconds 
    printf("Serial Execution time is: %0.3f milliseconds \n", time_taken * 1000);
    printf("Serial Execution time is: %0.3f milliseconds \n", elapsed_time * 1000);

    free(d_imageInSingle);
    free(d_imageOutSingle);

    /*
        End of serial algorithm.
    */



    // Allocate device memory for images
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Copy input image to device
    checkCudaErrors(hipMemcpy(d_imageIn, h_imageIn, datasize, hipMemcpyHostToDevice));

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Execute the kernel
    hipEventRecord(start);
    sharpen<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, cpp);
    getLastCudaError("sharpen() execution failed\n");
    hipEventRecord(stop);

    // Copy image back to host
    checkCudaErrors(hipMemcpy(h_imageOut, d_imageOut, datasize, hipMemcpyDeviceToHost));

    // Wait for the event to finish
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);

    // Retrieve output file type
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;
    while (token != NULL)
    {
        FileType = token;
        token = strtok(NULL, ".");
    }
    // Write output image to file
    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Release device memory
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));

    // Clean up the two events
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    // Release host memory
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}

/*

Execution times:

+-----------+------------+-------------+--------------+
| IMGSIZE   | Tcpu       | Tgpu        | S            |
+-----------+------------+-------------+--------------+
| 640x480   | 52.542ms   | 0.176ms     | 298.534      |
+-----------+------------+-------------+--------------+
| 800x600   | 79.712ms   | 0.235ms     | 339.200      |
+-----------+------------+-------------+--------------+
| 1600x900  | 261.934ms  | 0.559ms     | 468.576      |
+-----------+------------+-------------+--------------+
| 1920x1080 | 375.542ms  | 0.776ms     | 483.946      |
+-----------+------------+-------------+--------------+
| 3840x2160 | 1535.421ms | 2.935ms     | 523.142      |
+-----------+------------+-------------+--------------+


*/
