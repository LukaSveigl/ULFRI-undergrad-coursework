#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

/** The number of grayscale values in the image. */
#define GRAYLEVELS 256
/** The number of desired channels in image. */
#define DESIRED_NCHANNELS 1
/** The size of a CUDA block. */
#define BLOCK_SIZE 256

// UTILITY FUNCTIONS

/**
 * Finds the minimum in the cumulative distribution function.
 * @param cdf - The cumulative distribution function.
 * @return - The minimum.
 */
unsigned long findMin(unsigned int *cdf) {
    unsigned long min = 0;
    for (int i = 0; min == 0 && i < GRAYLEVELS; i++) {
		min = cdf[i];
    }
    return min;
}

/**
 * Scales the cumulative distribution function.
 * @param cdf       - The cumulative distribution function.
 * @param cdfmin    - The cumulative distribution function minimum.
 * @param imageSize - The image size.
 * @return - The scale.
 */
__device__ inline unsigned char scale(unsigned long cdf, unsigned long cdfmin, unsigned long imageSize) {
    float scale;
    scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    scale = round(scale * (float)(GRAYLEVELS-1));
    return (int)scale;
}

// HISTOGRAM EQUALIZATION ALGORITHM FUNCTIONS

/**
 * Creates the histogram for the input grayscale image.
 * @param image     - The grayscale image.
 * @param width     - The image width.
 * @param height    - The image height.
 * @param histogram - The computed histogram.
 */
__global__ void CalculateHistogram(unsigned char *image, int width, int height, unsigned int *histogram) { 
    __shared__ unsigned int shared_cache[GRAYLEVELS];

    // Clear the shared cache.
    if (threadIdx.x < GRAYLEVELS) {
        shared_cache[threadIdx.x] = 0;
    }

    __syncthreads();

    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (threadId < width * height) {
        // Atomic operations must be employed because the threads access
        // the same memory.
        atomicAdd(&(shared_cache[(int)image[threadId]]), 1);
        threadId += stride;
    }

    __syncthreads();

    if (threadIdx.x < GRAYLEVELS) {
        atomicAdd(&(histogram[threadIdx.x]), shared_cache[threadIdx.x]);
    }
}

/**
 * Calculates the cumulative distribution histogram using the Work-Efficient Sum Scan implementation provided
 * here: https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
 * @param histogram - The histogram.
 * @param cdf       - The cumulative distribution function.
 */
__global__ void CalculateCDF(unsigned int *histogram, unsigned int *cdf, int width, int height) {
    extern __shared__ float scan[2 * GRAYLEVELS];

    int threadId = threadIdx.x;
    int offset = 1;

    scan[2 * threadId] = histogram[2 * threadId];
    scan[2 * threadId + 1] = histogram[2 * threadId + 1];

    // Build sum in place.
    for (int d = GRAYLEVELS >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            scan[bi] += scan[ai];
        }

        offset *= 2;
    }

    if (threadId == 0) {
        scan[GRAYLEVELS - 1] = 0;
    }

    // Traverse down tree and build scan.
    for (int d = 1; d < GRAYLEVELS; d *= 2) {
        offset >>= 1;
        __syncthreads();

        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;

            float t = scan[ai];
            scan[ai] = scan[bi];
            scan[bi] += t;
        }
    }
    __syncthreads();

    // Store results of scan into cumulative distribution function.
    cdf[2 * threadId] = scan[2 * threadId];
    cdf[2 * threadId + 1] = scan[2 * threadId + 1];
}

/**
 * Equalizes the image with the histogram.
 * @param image_in  - The input image.
 * @param image_out - The output image.
 * @param width     - The image width.
 * @param height    - The image height.
 * @param cdf       - The cumulative distribution function.
 * @param cdfmin    - The minimum in the cumulative distribution function.
 */
__global__ void Equalize(unsigned char *image_in, unsigned char *image_out, int width, int height, unsigned int* cdf, unsigned long cdfmin) {
    unsigned long imageSize = width * height;
    //unsigned long cdfmin = findMin(cdf);

    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = stride = blockDim.x * gridDim.x;

    while (threadId < imageSize) {
        image_out[threadId] = scale(cdf[(int)image_in[threadId]], cdfmin, imageSize);
        threadId += stride;
    }
    __syncthreads();
}


int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("USAGE: %s input_image output_image\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);


	//
    // Read image from file.
    //

    int width, height, cpp;
    // Read only DESIRED_NCHANNELS channels from the input image.
    unsigned char *imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, DESIRED_NCHANNELS);
    if(imageIn == NULL) {
        printf("Error in loading the image\n");
        return 1;
    }
    const size_t datasize = height * width * cpp * sizeof(unsigned long);
    printf("Loaded image W = %d, H = %d, actual cpp = %d \n", width, height, cpp);
    
    int imageSize = width * height * cpp;
	//unsigned char *imageOut = (unsigned char *)malloc(height * width * sizeof(unsigned long));
    unsigned char *imageOut = (unsigned char *)malloc(datasize);



    //
    // Setup device variables.
    //

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;
    unsigned int *d_histogram;
    unsigned int *d_CDF;

    // Prepare memory for device. This entails allocating memory on the device, copying read
    // image to device, etc.
    checkCudaErrors(hipMalloc(&d_imageIn, width * height * sizeof(unsigned char)));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));
    checkCudaErrors(hipMemcpy(d_imageIn, imageIn, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&d_histogram, GRAYLEVELS * sizeof(unsigned int)));
    checkCudaErrors(hipMalloc(&d_CDF, GRAYLEVELS * sizeof(unsigned int)));

    // Set histogram and cdf to 0, would be better if it was done parallel.
    checkCudaErrors(hipMemset(d_histogram, 0, GRAYLEVELS * sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_CDF, 0, GRAYLEVELS * sizeof(unsigned int)));

    // Prepare CUDA events for time measurement.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Calculate the grid and block sizes.    
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(ceil(imageSize) / BLOCK_SIZE);


    //
    // Execute kernel.
    //

	// Create the histogram of grayscale values for the given image.
	CalculateHistogram<<<gridSize, blockSize>>>(d_imageIn, width, height, d_histogram);

    #ifdef DEBUG_HIST
        unsigned int *debug_hist = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
        checkCudaErrors(hipMemcpy(debug_hist, d_histogram, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost));

        for (int i = 0; i < GRAYLEVELS; i++) {
            printf("hist[%i]=%i\n", i, debug_hist[i]);
        }
    #endif

	// Calculate the cumulative distribution histogram using the Work-Efficient Sum Scan.
	CalculateCDF<<<1, GRAYLEVELS>>>(d_histogram, d_CDF, width, height);

    #ifdef DEBUG_CDF
        unsigned int *debug_cdf = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
        checkCudaErrors(hipMemcpy(debug_cdf, d_CDF, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost));

        for (int i = 0; i < GRAYLEVELS; i++) {
            printf("cdf[%i]=%i\n", i, debug_cdf[i]);
        }
    #endif

    // Compute the min in CDF on the CPU, as it only needs to be computed once. 
    unsigned int *CDF = (unsigned int*)malloc(GRAYLEVELS * sizeof(unsigned int));
    checkCudaErrors(hipMemcpy(CDF, d_CDF, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost));
    unsigned long cdfmin = findMin(CDF);
    free(CDF);

	// Equalize the image using the calculated cumulative distribution function.
	Equalize<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, d_CDF, cdfmin);

    //
    // Kernel finished.
    //


    checkCudaErrors(hipMemcpy(imageOut, d_imageOut, datasize, hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);


    //
    // Save image to file.
    //

    // Retrieve output file type.
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;

    while (token != NULL) {
        FileType = token;
        token = strtok(NULL, ".");
    }

    // Write output image to file.
    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    hipEventDestroy(start);
	hipEventDestroy(stop);


    /*
     * Free device and CPU memory.
     */

    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));
    checkCudaErrors(hipFree(d_histogram));
    checkCudaErrors(hipFree(d_CDF));

	free(imageIn);
    free(imageOut);

	return 0;
}



